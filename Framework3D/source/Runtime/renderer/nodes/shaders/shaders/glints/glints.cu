#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "../Optix/ShaderNameAbbre.h"
#include "glints_params.h"

inline unsigned GetLaunchID()
{
    uint3 launch_index = optixGetLaunchIndex();
    return launch_index.x;
}

__device__ float2 operator+(const float2& a, const float2& b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float2 operator/(const float2& a, const float b)
{
    return make_float2(a.x / b, a.y / b);
}

RGS(line)
{
    auto patch = params.patches[GetLaunchID()];

    auto uv_center = (patch.uv0 + patch.uv1 + patch.uv2 + patch.uv3) / 4.0f;

    float3 origin = make_float3(uv_center.x, uv_center.y, 100.0f);

    float3 dir = make_float3(0, 0, -1);

    optixTrace(
        params.handle,
        origin,
        dir,
        0,
        200.f,
        1.0,
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_NONE,
        0,
        1,
        0);
}

CHS(line)
{
}

MISS(line)
{
}

AHS(line)
{
    auto lineid = optixGetPrimitiveIndex();
    params.patch_line_pairs->Push({ lineid, GetLaunchID() });
    optixIgnoreIntersection();
}
