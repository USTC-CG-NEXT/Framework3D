#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>

#include "../Optix/ShaderNameAbbre.h"
#include "mesh_params.h"

inline unsigned GetLaunchID()
{
    uint3 launch_index = optixGetLaunchIndex();
    return launch_index.x;
}

__device__ float2 operator+(const float2& a, const float2& b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float2 operator/(const float2& a, const float b)
{
    return make_float2(a.x / b, a.y / b);
}

__device__ float2 operator/(const float2& a, const float2& b)
{
    return make_float2(a.x / b.x, a.y / b.y);
}

__device__ float3 operator+(const float3& a, const float3& b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator-(const float3& a, const float3& b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float3 normalize(const float3& v)
{
    float length = sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
    return make_float3(v.x / length, v.y / length, v.z / length);
}

__device__ float3 make_float3(const float4& a)
{
    return make_float3(a.x, a.y, a.z);
}

__device__ float4 make_float4(const float3& a, const float b)
{
    return make_float4(a.x, a.y, a.z, b);
}

__device__ float4 operator/=(float4& a, const float b)
{
    a.x /= b;
    a.y /= b;
    a.z /= b;
    a.w /= b;
    return a;
}

__device__ float4 operator/(const float4& a, const float b)
{
    return make_float4(a.x / b, a.y / b, a.z / b, a.w / b);
}

struct Payload {
    float2 uv;
    float3 corner0;
    float3 corner1;
    float3 corner2;
    unsigned hit;

    void set_self()
    {
        optixSetPayload_0(__float_as_uint(uv.x));
        optixSetPayload_1(__float_as_uint(uv.y));
        optixSetPayload_2(__float_as_uint(corner0.x));
        optixSetPayload_3(__float_as_uint(corner0.y));
        optixSetPayload_4(__float_as_uint(corner0.z));
        optixSetPayload_5(__float_as_uint(corner1.x));
        optixSetPayload_6(__float_as_uint(corner1.y));
        optixSetPayload_7(__float_as_uint(corner1.z));
        optixSetPayload_8(__float_as_uint(corner2.x));
        optixSetPayload_9(__float_as_uint(corner2.y));
        optixSetPayload_10(__float_as_uint(corner2.z));
        optixSetPayload_11(hit);
    }
};
#define Payload_As_Params(payload_name)                          \
    reinterpret_cast<unsigned int&>(payload_name.uv.x),          \
        reinterpret_cast<unsigned int&>(payload_name.uv.y),      \
        reinterpret_cast<unsigned int&>(payload_name.corner0.x), \
        reinterpret_cast<unsigned int&>(payload_name.corner0.y), \
        reinterpret_cast<unsigned int&>(payload_name.corner0.z), \
        reinterpret_cast<unsigned int&>(payload_name.corner1.x), \
        reinterpret_cast<unsigned int&>(payload_name.corner1.y), \
        reinterpret_cast<unsigned int&>(payload_name.corner1.z), \
        reinterpret_cast<unsigned int&>(payload_name.corner2.x), \
        reinterpret_cast<unsigned int&>(payload_name.corner2.y), \
        reinterpret_cast<unsigned int&>(payload_name.corner2.z), \
        payload_name.hit

__device__ void calculateRayParameters(
    const uint3& launch_index,
    const uint3& launch_dimensions,
    float bias_x,
    float bias_y,
    float3& origin,
    float3& direction)
{
    float2 pixel_position_f =
        make_float2(launch_index.x + bias_x, launch_index.y + bias_y);
    float2 uv = pixel_position_f /
                make_float2(launch_dimensions.x, launch_dimensions.y);
    float4 clip_pos =
        make_float4(uv.x * 2.0f - 1.0f, uv.y * 2.0f - 1.0f, 1.0f, 1.0f);

    auto clipToView = mesh_params.viewToClip.get_inverse();
    auto viewToWorld = mesh_params.worldToView.get_inverse();

    float4 view_pos = clipToView * clip_pos;
    view_pos /= view_pos.w;

    auto view_space_direction = (make_float3(view_pos) - make_float3(0, 0, 0));

    origin = make_float3(viewToWorld * make_float4(0, 0, 0, 1));
    direction = normalize(
        make_float3(viewToWorld * make_float4(view_space_direction, 0)));
}

__device__ void traceRayAndSetPayload(
    const uint3& launch_index,
    const uint3& launch_dimensions,
    float bias_x,
    float bias_y,
    float3& origin,
    float3& direction,
    Payload& payload)
{
    calculateRayParameters(
        launch_index, launch_dimensions, bias_x, bias_y, origin, direction);

    optixTrace(
        mesh_params.handle,
        origin,
        direction,
        0.0f,
        1e5f,
        1.0f,
        OptixVisibilityMask(255),
        unsigned(OPTIX_RAY_FLAG_NONE),
        unsigned(0),
        unsigned(1),
        unsigned(0),
        Payload_As_Params(payload));
}

RGS(mesh)
{
    uint3 launch_index = optixGetLaunchIndex();
    uint3 launch_dimensions = optixGetLaunchDimensions();

    float3 origin;
    float3 direction;

    Payload payload;
    payload.hit = false;

    traceRayAndSetPayload(
        launch_index,
        launch_dimensions,
        0.5f,
        0.5f,
        origin,
        direction,
        payload);

    if (launch_index.x == 0 && launch_index.y == 0) {
        printf("origin: %f %f %f\n", origin.x, origin.y, origin.z);
    }

    if (payload.hit) {
        Patch patch;

        traceRayAndSetPayload(
            launch_index, launch_dimensions, 0, 0, origin, direction, payload);
        patch.uv0 = payload.uv;

        traceRayAndSetPayload(
            launch_index, launch_dimensions, 1, 0, origin, direction, payload);
        patch.uv1 = payload.uv;

        traceRayAndSetPayload(
            launch_index, launch_dimensions, 1, 1, origin, direction, payload);
        patch.uv2 = payload.uv;

        traceRayAndSetPayload(
            launch_index, launch_dimensions, 0, 1, origin, direction, payload);
        patch.uv3 = payload.uv;

        auto id = mesh_params.append_buffer->Push(patch);
        mesh_params.corners[id].v0 = payload.corner0;
        mesh_params.corners[id].v1 = payload.corner1;
        mesh_params.corners[id].v2 = payload.corner2;
        mesh_params.pixel_targets[id] =
            make_int2(launch_index.x, launch_index.y);
    }
}

struct Vertex {
    float3 position;
    float2 uv;
};

__device__ float2 operator*(const float2& a, const float b)
{
    return make_float2(a.x * b, a.y * b);
}

CHS(mesh)
{
    Payload payload;
    auto primitiveid = optixGetPrimitiveIndex();
    uint3 indices = reinterpret_cast<uint3*>(mesh_params.indices)[primitiveid];

    auto vertex0 = reinterpret_cast<Vertex*>(mesh_params.vertices)[indices.x];
    auto vertex1 = reinterpret_cast<Vertex*>(mesh_params.vertices)[indices.y];
    auto vertex2 = reinterpret_cast<Vertex*>(mesh_params.vertices)[indices.z];

    payload.corner0 = vertex0.position;
    payload.corner1 = vertex1.position;
    payload.corner2 = vertex2.position;

    auto barycentric = optixGetTriangleBarycentrics();

    payload.uv = vertex0.uv * (1.0f - barycentric.x - barycentric.y) +
                 vertex1.uv * barycentric.x + vertex2.uv * barycentric.y;

    payload.hit = true;
    payload.set_self();
}

MISS(mesh)
{
}

AHS(mesh)
{
}
