#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>

#include "../Optix/ShaderNameAbbre.h"
#include "mesh_params.h"

inline unsigned GetLaunchID()
{
    uint3 launch_index = optixGetLaunchIndex();
    return launch_index.x;
}

__device__ float2 operator+(const float2& a, const float2& b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float2 operator/(const float2& a, const float b)
{
    return make_float2(a.x / b, a.y / b);
}

__device__ float2 operator/(const float2& a, const float2& b)
{
    return make_float2(a.x / b.x, a.y / b.y);
}

__device__ float3 operator+(const float3& a, const float3& b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator-(const float3& a, const float3& b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float3 normalize(const float3& v)
{
    float length = sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
    return make_float3(v.x / length, v.y / length, v.z / length);
}

__device__ float3 make_float3(const float4& a)
{
    return make_float3(a.x, a.y, a.z);
}

__device__ float4 make_float4(const float3& a, const float b)
{
    return make_float4(a.x, a.y, a.z, b);
}

__device__ float4 operator/=(float4& a, const float b)
{
    a.x /= b;
    a.y /= b;
    a.z /= b;
    a.w /= b;
    return a;
}

__device__ float4 operator/(const float4& a, const float b)
{
    return make_float4(a.x / b, a.y / b, a.z / b, a.w / b);
}

struct Payload {
    float2 uv;
    float4x4 worldToUV;
    unsigned hit;

    void set_self()
    {
        optixSetPayload_0(__float_as_uint(uv.x));
        optixSetPayload_1(__float_as_uint(uv.y));
        optixSetPayload_2(__float_as_uint(worldToUV.m[0][0]));
        optixSetPayload_3(__float_as_uint(worldToUV.m[0][1]));
        optixSetPayload_4(__float_as_uint(worldToUV.m[0][2]));
        optixSetPayload_5(__float_as_uint(worldToUV.m[0][3]));
        optixSetPayload_6(__float_as_uint(worldToUV.m[1][0]));
        optixSetPayload_7(__float_as_uint(worldToUV.m[1][1]));
        optixSetPayload_8(__float_as_uint(worldToUV.m[1][2]));
        optixSetPayload_9(__float_as_uint(worldToUV.m[1][3]));
        optixSetPayload_10(__float_as_uint(worldToUV.m[2][0]));
        optixSetPayload_11(__float_as_uint(worldToUV.m[2][1]));
        optixSetPayload_12(__float_as_uint(worldToUV.m[2][2]));
        optixSetPayload_13(__float_as_uint(worldToUV.m[2][3]));
        optixSetPayload_14(__float_as_uint(worldToUV.m[3][0]));
        optixSetPayload_15(__float_as_uint(worldToUV.m[3][1]));
        optixSetPayload_16(__float_as_uint(worldToUV.m[3][2]));
        optixSetPayload_17(__float_as_uint(worldToUV.m[3][3]));
        optixSetPayload_18(hit);
    }
};

#define Payload_As_Params(payload_name)                                  \
    reinterpret_cast<unsigned int&>(payload_name.uv.x),                  \
        reinterpret_cast<unsigned int&>(payload_name.uv.y),              \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[0][0]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[0][1]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[0][2]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[0][3]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[1][0]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[1][1]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[1][2]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[1][3]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[2][0]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[2][1]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[2][2]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[2][3]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[3][0]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[3][1]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[3][2]), \
        reinterpret_cast<unsigned int&>(payload_name.worldToUV.m[3][3]), \
        payload_name.hit

__device__ void calculateRayParameters(
    const uint3& launch_index,
    const uint3& launch_dimensions,
    float bias_x,
    float bias_y,
    float3& origin,
    float3& direction)
{
    float2 pixel_position_f =
        make_float2(launch_index.x + bias_x, launch_index.y + bias_y);
    float2 uv = pixel_position_f /
                make_float2(launch_dimensions.x, launch_dimensions.y);
    float4 clip_pos =
        make_float4(uv.x * 2.0f - 1.0f, uv.y * 2.0f - 1.0f, 1.0f, 1.0f);

    auto clipToView = mesh_params.viewToClip.get_inverse();
    auto viewToWorld = mesh_params.worldToView.get_inverse();

    float4 view_pos = clipToView * clip_pos;
    view_pos /= view_pos.w;

    auto view_space_direction = (make_float3(view_pos) - make_float3(0, 0, 0));

    origin = make_float3(viewToWorld * make_float4(0, 0, 0, 1));
    direction = normalize(
        make_float3(viewToWorld * make_float4(view_space_direction, 0)));
}

__device__ void traceRayAndSetPayload(
    const uint3& launch_index,
    const uint3& launch_dimensions,
    float bias_x,
    float bias_y,
    float3& origin,
    float3& direction,
    Payload& payload)
{
    calculateRayParameters(
        launch_index, launch_dimensions, bias_x, bias_y, origin, direction);

    optixTrace(
        mesh_params.handle,
        origin,
        direction,
        0.0f,
        1e5f,
        1.0f,
        OptixVisibilityMask(255),
        unsigned(OPTIX_RAY_FLAG_NONE),
        unsigned(0),
        unsigned(1),
        unsigned(0),
        Payload_As_Params(payload));
}

RGS(mesh)
{
    uint3 launch_index = optixGetLaunchIndex();
    uint3 launch_dimensions = optixGetLaunchDimensions();

    float3 origin;
    float3 direction;

    Payload payload;
    payload.hit = false;

    traceRayAndSetPayload(
        launch_index,
        launch_dimensions,
        0.5f,
        0.5f,
        origin,
        direction,
        payload);

    if (payload.hit) {
        Patch patch;

        traceRayAndSetPayload(
            launch_index, launch_dimensions, 0, 0, origin, direction, payload);
        patch.uv0 = payload.uv;

        traceRayAndSetPayload(
            launch_index, launch_dimensions, 1, 0, origin, direction, payload);
        patch.uv1 = payload.uv;

        traceRayAndSetPayload(
            launch_index, launch_dimensions, 1, 1, origin, direction, payload);
        patch.uv2 = payload.uv;

        traceRayAndSetPayload(
            launch_index, launch_dimensions, 0, 1, origin, direction, payload);
        patch.uv3 = payload.uv;

        auto id = mesh_params.append_buffer->Push(patch);
        mesh_params.worldToUV[id] = payload.worldToUV;
        mesh_params.pixel_targets[id] =
            make_int2(launch_index.x, launch_index.y);
    }
}

__device__ float2 operator*(const float2& a, const float b)
{
    return make_float2(a.x * b, a.y * b);
}

__device__ float3 cross(const float3& a, const float3& b)
{
    return make_float3(
        a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}
__device__ float3 operator-(const float3& a)
{
    return make_float3(-a.x, -a.y, -a.z);
}
struct Vertex {
    float pos_x, pos_y, pos_z;
    float u, v;
};

CHS(mesh)
{
    Payload payload;
    auto primitiveid = optixGetPrimitiveIndex();
    uint3 indices = reinterpret_cast<uint3*>(mesh_params.indices)[primitiveid];

    auto vertex0 = reinterpret_cast<Vertex*>(mesh_params.vertices)[indices.x];
    auto vertex1 = reinterpret_cast<Vertex*>(mesh_params.vertices)[indices.y];
    auto vertex2 = reinterpret_cast<Vertex*>(mesh_params.vertices)[indices.z];

    float2 uv0 = make_float2(vertex0.u, vertex0.v);
    float2 uv1 = make_float2(vertex1.u, vertex1.v);
    float2 uv2 = make_float2(vertex2.u, vertex2.v);

    float3 pos0 = make_float3(vertex0.pos_x, vertex0.pos_y, vertex0.pos_z);
    float3 pos1 = make_float3(vertex1.pos_x, vertex1.pos_y, vertex1.pos_z);
    float3 pos2 = make_float3(vertex2.pos_x, vertex2.pos_y, vertex2.pos_z);

    float3 normal = normalize(cross(pos1 - pos0, pos2 - pos0));

    float4x4 target(
        make_float4(uv0.x, uv0.y, 0.f, 1.f),
        make_float4(uv1.x, uv1.y, 0.f, 1.f),
        make_float4(uv2.x, uv2.y, 0.f, 1.f),
        make_float4(0, 0, 1, 0));
    float4x4 points(
        make_float4(pos0.x, pos0.y, pos0.z, 1),
        make_float4(pos1.x, pos1.y, pos1.z, 1),
        make_float4(pos2.x, pos2.y, pos2.z, 1),
        make_float4(normal, 0));

    payload.worldToUV = target * points.get_inverse();

    auto barycentric = optixGetTriangleBarycentrics();

    payload.uv = uv0 * (1.0f - barycentric.x - barycentric.y) +
                 uv1 * barycentric.x + uv2 * barycentric.y;

    payload.hit = true;
    payload.set_self();
}

MISS(mesh)
{
}

AHS(mesh)
{
}
