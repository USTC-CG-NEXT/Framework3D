#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "../Optix/ShaderNameAbbre.h"
#include "params.h"

inline unsigned GetLaunchID()
{
    uint3 launch_index = optixGetLaunchIndex();
    return launch_index.x;
}

__device__ float2 operator+(const float2& a, const float2& b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float2 operator/(const float2& a, const float b)
{
    return make_float2(a.x / b, a.y / b);
}

__device__ float2 operator/(const float2& a, const float2& b)
{
    return make_float2(a.x / b.x, a.y / b.y);
}

__device__ float3 operator+(const float3& a, const float3& b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator-(const float3& a, const float3& b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float3 normalize(const float3& v)
{
    float length = sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
    return make_float3(v.x / length, v.y / length, v.z / length);
}

__device__ float3 make_float3(const float4& a)
{
    return make_float3(a.x, a.y, a.z);
}

__device__ float4 operator/=(float4& a, const float b)
{
    a.x /= b;
    a.y /= b;
    a.z /= b;
    a.w /= b;
    return a;
}

RGS(mesh)
{
    uint3 launch_index = optixGetLaunchIndex();
    uint3 launch_dimensions = optixGetLaunchDimensions();

    float2 pixel_position_f = make_float2(launch_index.x, launch_index.y);

    float2 uv = pixel_position_f /
                make_float2(launch_dimensions.x, launch_dimensions.y);
    float4 clip_pos =
        make_float4(uv.x * 2.0f - 1.0f, uv.y * 2.0f - 1.0f, 1.0f, 1.0f);

    auto clipToWorld = mesh_params.worldToClip.get_inverse();

    float4 view_pos = clipToWorld * clip_pos;
    view_pos /= view_pos.w;

    float3 origin = make_float3(0, 0, 0);

    float3 camera_right = make_float3(1, 0, 0);
    float3 camera_up = make_float3(0, 1, 0);

    float3 direction = normalize(make_float3(view_pos) - origin);

    optixTrace(
        mesh_params.handle,
        origin,
        direction,
        0.0f,
        1e5f,
        1.0f,
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_NONE,
        0,
        1,
        0);
}

CHS(mesh)
{
}

MISS(mesh)
{
}

AHS(mesh)
{
}
