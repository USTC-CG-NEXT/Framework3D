#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "../Optix/ShaderNameAbbre.h"
#include "params.h"

inline unsigned GetLaunchID()
{
    uint3 launch_index = optixGetLaunchIndex();
    return launch_index.x;
}

__device__ float2 operator+(const float2& a, const float2& b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float2 operator/(const float2& a, const float b)
{
    return make_float2(a.x / b, a.y / b);
}

RGS(mesh)
{
    auto worldToClip = mesh_params.worldToClip;
    auto clipToWorld = worldToClip.get_inverse();

    uint3 dispatch_index = optixGetLaunchIndex();
    uint3 dispatch_dim = optixGetLaunchDimensions();

    float2 uv = make_float2(
        (dispatch_index.x + 0.5f) / dispatch_dim.x,
        (dispatch_index.y + 0.5f) / dispatch_dim.y);

    float3 origin = make_float3(0, 0, 0);
    float4 clip_rayend = make_float4(uv * 2.0f - 1.0f, 0.0f, 1.0f);

    float4 world_rayend = clipToWorld * clip;

    // optixTrace(
    //     params.handle,
    //     origin,
    //     dir,
    //     0,
    //     1e5f,
    //     1.0,
    //     OptixVisibilityMask(255),
    //     OPTIX_RAY_FLAG_NONE,
    //     0,
    //     1,
    //     0);
}

CHS(mesh)
{
}

MISS(mesh)
{
}

AHS(mesh)
{
}
