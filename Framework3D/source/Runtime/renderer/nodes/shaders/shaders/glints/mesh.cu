#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>

#include "../Optix/ShaderNameAbbre.h"
#include "mesh_params.h"

inline unsigned GetLaunchID()
{
    uint3 launch_index = optixGetLaunchIndex();
    return launch_index.x;
}

__device__ float2 operator+(const float2& a, const float2& b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float2 operator/(const float2& a, const float b)
{
    return make_float2(a.x / b, a.y / b);
}

__device__ float2 operator/(const float2& a, const float2& b)
{
    return make_float2(a.x / b.x, a.y / b.y);
}

__device__ float3 operator+(const float3& a, const float3& b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator-(const float3& a, const float3& b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float3 normalize(const float3& v)
{
    float length = sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
    return make_float3(v.x / length, v.y / length, v.z / length);
}

__device__ float3 make_float3(const float4& a)
{
    return make_float3(a.x, a.y, a.z);
}

__device__ float4 make_float4(const float3& a, const float b)
{
    return make_float4(a.x, a.y, a.z, b);
}

__device__ float4 operator/=(float4& a, const float b)
{
    a.x /= b;
    a.y /= b;
    a.z /= b;
    a.w /= b;
    return a;
}

__device__ float4 operator/(const float4& a, const float b)
{
    return make_float4(a.x / b, a.y / b, a.z / b, a.w / b);
}

struct Payload {
    float2 uv;
    float3 corner0;
    float3 corner1;
    float3 corner2;
    unsigned hit;

    void set_self()
    {
        optixSetPayload_0(__float_as_uint(uv.x));
        optixSetPayload_1(__float_as_uint(uv.y));
        optixSetPayload_2(__float_as_uint(corner0.x));
        optixSetPayload_3(__float_as_uint(corner0.y));
        optixSetPayload_4(__float_as_uint(corner0.z));
        optixSetPayload_5(__float_as_uint(corner1.x));
        optixSetPayload_6(__float_as_uint(corner1.y));
        optixSetPayload_7(__float_as_uint(corner1.z));
        optixSetPayload_8(__float_as_uint(corner2.x));
        optixSetPayload_9(__float_as_uint(corner2.y));
        optixSetPayload_10(__float_as_uint(corner2.z));
        optixSetPayload_11(hit);
    }
};
#define Payload_As_Params(payload_name)                          \
    reinterpret_cast<unsigned int&>(payload_name.uv.x),          \
        reinterpret_cast<unsigned int&>(payload_name.uv.y),      \
        reinterpret_cast<unsigned int&>(payload_name.corner0.x), \
        reinterpret_cast<unsigned int&>(payload_name.corner0.y), \
        reinterpret_cast<unsigned int&>(payload_name.corner0.z), \
        reinterpret_cast<unsigned int&>(payload_name.corner1.x), \
        reinterpret_cast<unsigned int&>(payload_name.corner1.y), \
        reinterpret_cast<unsigned int&>(payload_name.corner1.z), \
        reinterpret_cast<unsigned int&>(payload_name.corner2.x), \
        reinterpret_cast<unsigned int&>(payload_name.corner2.y), \
        reinterpret_cast<unsigned int&>(payload_name.corner2.z), \
        payload_name.hit

__device__ void calculateRayParameters(
    const uint3& launch_index,
    const uint3& launch_dimensions,
    float bias_x,
    float bias_y,
    float3& origin,
    float3& direction)
{
    float2 pixel_position_f =
        make_float2(launch_index.x + bias_x, launch_index.y + bias_y);
    float2 uv = pixel_position_f /
                make_float2(launch_dimensions.x, launch_dimensions.y);
    float4 clip_pos =
        make_float4(uv.x * 2.0f - 1.0f, uv.y * 2.0f - 1.0f, 1.0f, 1.0f);

    origin = make_float3(0, 0, 0);
    direction = normalize(make_float3(clip_pos) - origin);

    auto clipToWorld = mesh_params.worldToClip.get_inverse();
    float4 world_origin = clipToWorld * make_float4(0, 0, 0, 1);
    origin = make_float3(world_origin / world_origin.w);
    direction = make_float3(clipToWorld * make_float4(direction, 0.f));
    direction = normalize(direction);
}

RGS(mesh)
{
    uint3 launch_index = optixGetLaunchIndex();
    uint3 launch_dimensions = optixGetLaunchDimensions();

    float bias_x = 0.5f;
    float bias_y = 0.5f;

    float3 origin;
    float3 direction;

    Payload payload;
    payload.hit = false;

    calculateRayParameters(
        launch_index, launch_dimensions, bias_x, bias_y, origin, direction);

    optixTrace(
        mesh_params.handle,
        origin,
        direction,
        0.0f,
        1e5f,
        1.0f,
        OptixVisibilityMask(255),
        unsigned(OPTIX_RAY_FLAG_NONE),
        unsigned(0),
        unsigned(1),
        unsigned(0),
        Payload_As_Params(payload));

    if (payload.hit) {
        Patch patch;

        calculateRayParameters(
            launch_index, launch_dimensions, 0, 0, origin, direction);

        optixTrace(
            mesh_params.handle,
            origin,
            direction,
            0.0f,
            1e5f,
            1.0f,
            OptixVisibilityMask(255),
            unsigned(OPTIX_RAY_FLAG_NONE),
            unsigned(0),
            unsigned(1),
            unsigned(0),
            Payload_As_Params(payload));
        patch.uv0 = payload.uv;

        calculateRayParameters(
            launch_index, launch_dimensions, 1, 0, origin, direction);
        optixTrace(
            mesh_params.handle,
            origin,
            direction,
            0.0f,
            1e5f,
            1.0f,
            OptixVisibilityMask(255),
            unsigned(OPTIX_RAY_FLAG_NONE),
            unsigned(0),
            unsigned(1),
            unsigned(0),
            Payload_As_Params(payload));
        patch.uv1 = payload.uv;

        calculateRayParameters(
            launch_index, launch_dimensions, 1, 1, origin, direction);

        optixTrace(
            mesh_params.handle,
            origin,
            direction,
            0.0f,
            1e5f,
            1.0f,
            OptixVisibilityMask(255),
            unsigned(OPTIX_RAY_FLAG_NONE),
            unsigned(0),
            unsigned(1),
            unsigned(0),
            Payload_As_Params(payload));

        patch.uv2 = payload.uv;
        calculateRayParameters(
            launch_index, launch_dimensions, 0, 1, origin, direction);

        optixTrace(
            mesh_params.handle,
            origin,
            direction,
            0.0f,
            1e5f,
            1.0f,
            OptixVisibilityMask(255),
            unsigned(OPTIX_RAY_FLAG_NONE),
            unsigned(0),
            unsigned(1),
            unsigned(0),
            Payload_As_Params(payload));
        patch.uv3 = payload.uv;

        auto id = mesh_params.append_buffer->Push(patch);
        mesh_params.corners[id].v0 = payload.corner0;
        mesh_params.corners[id].v1 = payload.corner1;
        mesh_params.corners[id].v2 = payload.corner2;
        mesh_params.pixel_targets[id] =
            make_int2(launch_index.x, launch_index.y);
    }
}

CHS(mesh)
{
    Payload payload;
    auto primitiveid = optixGetPrimitiveIndex();
    uint3 indices = reinterpret_cast<uint3*>(mesh_params.indices)[primitiveid];
    payload.corner0 =
        reinterpret_cast<float3*>(mesh_params.vertices)[indices.x];
    payload.corner1 =
        reinterpret_cast<float3*>(mesh_params.vertices)[indices.y];
    payload.corner2 =
        reinterpret_cast<float3*>(mesh_params.vertices)[indices.z];

    payload.uv = optixGetTriangleBarycentrics();

    payload.hit = true;
    payload.set_self();
}

MISS(mesh)
{
}

AHS(mesh)
{
}
