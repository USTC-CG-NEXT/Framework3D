#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "../Optix/ShaderNameAbbre.h"
#include "params.h"

inline unsigned GetLaunchID()
{
    uint3 launch_index = optixGetLaunchIndex();
    return launch_index.x;
}

__device__ float2 operator+(const float2& a, const float2& b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float2 operator/(const float2& a, const float b)
{
    return make_float2(a.x / b, a.y / b);
}

RGS(mesh)
{
    auto rays = mesh_params.rays[GetLaunchID()];

    //auto uv_center = (patch.uv0 + patch.uv1 + patch.uv2 + patch.uv3) / 4.0f;

    //float3 origin = make_float3(uv_center.x, uv_center.y, 10000.0f);

    //float3 dir = make_float3(0, 0, -1);

    //optixTrace(
    //    params.handle,
    //    origin,
    //    dir,
    //    0,
    //    1e5f,
    //    1.0,
    //    OptixVisibilityMask(255),
    //    OPTIX_RAY_FLAG_NONE,
    //    0,
    //    1,
    //    0);
}

CHS(mesh)
{

}

MISS(mesh)
{
}

AHS(mesh)
{
}
