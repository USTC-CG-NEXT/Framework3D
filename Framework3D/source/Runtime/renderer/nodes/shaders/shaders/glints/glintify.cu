#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "glintify/stroke.h"


#include "../Optix/ShaderNameAbbre.h"
#include "glintify/glintify_params.h"

inline unsigned GetLaunchID()
{
    uint3 launch_index = optixGetLaunchIndex();
    return launch_index.x;
}

__device__ float2 operator+(const float2& a, const float2& b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float2 operator/(const float2& a, const float b)
{
    return make_float2(a.x / b, a.y / b);
}

float3 glm_to_float3(const glm::vec3& v)
{
    return make_float3(v.x, v.y, v.z);
}

RGS(mesh_glintify)
{
    auto id = GetLaunchID();
    auto stroke = params.strokes[id];

    auto camera_move_range = params.camera_move_range;

    constexpr unsigned sample_count = 256;

    auto camera_left = params.camera_position;
    camera_left.x += camera_move_range.x;

    auto camera_right = params.camera_position;
    camera_right.x += camera_move_range.y;

    for (int i = 0; i < sample_count; i++) {
        auto t = static_cast<float>(i) / (sample_count - 1);
        auto test_cam_pos = camera_left * (1 - t) + camera_right * t;

        auto dir = stroke.virtual_point_position - test_cam_pos;

        unsigned occluded = 0;
        optixTrace(
            params.handle,
            glm_to_float3(test_cam_pos),
            glm_to_float3(dir),
            0.0f,
            1.f,
            0.0f,
            OptixVisibilityMask(255),
            OPTIX_RAY_FLAG_NONE,
            0,
            1,
            0,
            occluded);
    }

    //auto tangent_vpt =
    //    stroke->world_to_tangent_point(stroke->virtual_point_position);

    //auto tangent_camera_left = stroke->world_to_tangent_point(camera_left);

    //auto tangent_camera_right = stroke->world_to_tangent_point(camera_right);

    //glm::vec2 on_image_left = (tangent_vpt - tangent_camera_left) *
    //                              (0 - tangent_camera_left.z) /
    //                              (tangent_vpt.z - tangent_camera_left.z) +
    //                          tangent_camera_left;

    //glm::vec2 on_image_right = (tangent_vpt - tangent_camera_right) *
    //                               (0 - tangent_camera_right.z) /
    //                               (tangent_vpt.z - tangent_camera_right.z) +
    //                           tangent_camera_right;

    //for (int i = 0; i < sample_count; i++) {
    //    auto t = static_cast<float>(i) / (sample_count - 1);
    //    auto on_image = on_image_left * (1 - t) + on_image_right * t;
    //}

    //stroke->range_count = 1;
    //stroke->range[0] = std::make_pair(on_image_left, on_image_right);
}

CHS(mesh_glintify)
{
    unsigned hit = 1;
    optixSetPayload_0(hit);
    printf("hit\n");

}
MISS(mesh_glintify)
{
}
