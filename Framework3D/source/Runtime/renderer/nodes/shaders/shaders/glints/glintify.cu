#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "../Optix/ShaderNameAbbre.h"
#include "glintify/glintify_params.h"
#include "glintify/stroke.h"

inline unsigned GetLaunchID()
{
    uint3 launch_index = optixGetLaunchIndex();
    return launch_index.x;
}

__device__ float2 operator+(const float2& a, const float2& b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float2 operator/(const float2& a, const float b)
{
    return make_float2(a.x / b, a.y / b);
}

float3 glm_to_float3(const glm::vec3& v)
{
    return make_float3(v.x, v.y, v.z);
}

RGS(mesh_glintify)
{
    auto id = GetLaunchID();
    auto stroke = params.strokes[id];

    auto camera_move_range = params.camera_move_range;

    constexpr unsigned sample_count = 1024;

    auto camera_left = params.camera_position;
    camera_left.x += camera_move_range.x;


    auto camera_right = params.camera_position;
    camera_right.x += camera_move_range.y;

    auto tangent_vpt =
        stroke->world_to_tangent_point(stroke->virtual_point_position);

    unsigned current_range = 0;

    bool taping = false;
    glm::vec2 on_image;

    for (int i = 0; i < sample_count; i++) {
        auto t = static_cast<float>(i) / (sample_count - 1);
        auto test_cam_pos =
            camera_left * (1 - t) + camera_right * t;


        auto dir = stroke->virtual_point_position - test_cam_pos;

        unsigned occluded = 0;
        optixTrace(
            params.handle,
            glm_to_float3(test_cam_pos),
            glm_to_float3(dir),
            0.0f,
            1.f,
            0.0f,
            OptixVisibilityMask(255),
            OPTIX_RAY_FLAG_NONE,
            0,
            1,
            0,
            occluded);

        //occluded = 0;
         
        bool start_taping = !taping && !occluded;
        bool end_taping = taping && occluded;
        
        test_cam_pos = stroke->world_to_tangent_point(test_cam_pos);


        on_image = (tangent_vpt - test_cam_pos) * (0 - test_cam_pos.z) /
                       (tangent_vpt.z - test_cam_pos.z) +
                   test_cam_pos;

        if (start_taping) {
            taping = true;
            stroke->range[current_range].first = on_image;
        }

        if (end_taping) {
            taping = false;
            stroke->range[current_range].second = on_image;

            current_range++;
        }
    }

    if (taping) {
        stroke->range[current_range].second = on_image;
        current_range++;
    }

    stroke->range_count = current_range;

    // camera_left = params.camera_position;
    //         camera_left.x += camera_move_range.x;
    //         auto tangent_camera_left =
    //             stroke->world_to_tangent_point(camera_left);

    //        camera_right = params.camera_position;
    //        camera_right.x += camera_move_range.y;

    //        auto tangent_camera_right =
    //            stroke->world_to_tangent_point(camera_right);

    //        glm::vec2 on_image_left =
    //            (tangent_vpt - tangent_camera_left) *
    //                (0 - tangent_camera_left.z) /
    //                (tangent_vpt.z - tangent_camera_left.z) +
    //            tangent_camera_left;

    //        glm::vec2 on_image_right =
    //            (tangent_vpt - tangent_camera_right) *
    //                (0 - tangent_camera_right.z) /
    //                (tangent_vpt.z - tangent_camera_right.z) +
    //            tangent_camera_right;

    //        if (on_image_left.x > on_image_right.x) {
    //            auto temp = on_image_left;
    //            on_image_left = on_image_right;
    //            on_image_right = temp;
    //        }

    //        stroke->range_count = 1;
    //        stroke->range[0] = cuda::std::make_pair(on_image_left,
    //        on_image_right);
}

CHS(mesh_glintify)
{
    unsigned hit = 1;
    optixSetPayload_0(hit);
}
MISS(mesh_glintify)
{
}
