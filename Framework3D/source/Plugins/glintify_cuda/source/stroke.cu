#include "hip/hip_runtime.h"

#include <glm/glm.hpp>

#include "glintify/stroke.h"

USTC_CG_NAMESPACE_OPEN_SCOPE
namespace stroke {

// Another question would be how to consider the luminance? the shading?
// By controlling the density of the scratches.
// But how does that mean exactly?
HOST_DEVICE glm::vec2 Stroke::eval_required_direction(
    glm::vec2 uv_space_pos,
    glm::vec3 light_pos)
{
    auto uv_space_vpt_pos = world_to_tangent_point(virtual_point_position);

    glm::vec3 tangent_space_cam_dir =
        uv_space_vpt_pos - glm::vec3(uv_space_pos, 0);
    if (uv_space_vpt_pos.z > 0) {
        tangent_space_cam_dir *= -1;
    }

    glm::vec3 tangent_space_light_dir =
        world_to_tangent_point(light_pos) - glm::vec3(uv_space_pos, 0);

    glm::vec<3, float> half_vec = glm::normalize(
        0.5f * (glm::normalize(tangent_space_cam_dir) +
                glm::normalize(tangent_space_light_dir)));

    return glm::vec2(-half_vec.y, half_vec.x);
}

HOST_DEVICE glm::vec2 same_direction(glm::vec2 vec, glm::vec2 reference)
{
    if (glm::dot(vec, reference) < 0) {
        return -vec;
    }
    return vec;
}

HOST_DEVICE void Stroke::calc_scratch(int scratch_index, glm::vec3 light_pos)
{
    scratch_count = MAX_SCRATCH_COUNT;

    auto tangent_space_light_pos = world_to_tangent_point(light_pos);

    float half_stroke_width = stroke_width / 2.0f;

    unsigned valid_sample_count = 0;

    glm::vec2 center_point;

    center_point.y = range[0].first.y;

    auto uv_vpt = world_to_tangent_point(virtual_point_position);

    uv_vpt.y = 2.0f * center_point.y - uv_vpt.y;

    glm::vec2 that_direction = uv_vpt - tangent_space_light_pos;
    center_point.x = tangent_space_light_pos.x +
                     (center_point.y - tangent_space_light_pos.y) *
                         that_direction.x / that_direction.y;

    auto vertical_movement =
        glm::vec2(0, 1) * stroke_width *
        (float(scratch_index / 2 + 0.5f) / float(MAX_SCRATCH_COUNT) - 0.25f) *
        90.0f;

    auto pos = center_point + glm::vec2(0.0001, 0) +
               glm::vec2(-1, 0) * float(scratch_index + 0.1f) /
                   float(MAX_SCRATCH_COUNT);

    pos = center_point + vertical_movement;

    glm::vec2 old_dir;

    for (int i = 0; i < SAMPLE_POINT_COUNT; ++i) {
        scratches[scratch_index].should_begin_new_line_mask[i] = false;
    }

    for (int i = 0; i < TEST_STEP_COUNT; ++i) {
        auto dir = eval_required_direction(pos, light_pos);

        if (i == 0) {
            auto scratch_going_right = dir.x > 0;
            if (!scratch_going_right) {
                dir *= -1;
            }
            bool other_way = scratch_index % 2 == 1;
            if (other_way) {
                dir *= -1;
            }
        }
        else {
            dir = same_direction(dir, old_dir);
        }

        old_dir = dir;

        if (std::abs(dir.y) > 0.999) {
            break;
        }

        auto step = 2.0f / float(TEST_STEP_COUNT);
        scratches[scratch_index].sample_point[valid_sample_count] = pos;

        auto temp_pos = pos;

        constexpr int substep_count = 10;

        auto sub_step = step / substep_count;

        for (int substep = 0; substep < substep_count; ++substep) {
            temp_pos += dir * sub_step;

            dir = eval_required_direction(temp_pos, light_pos);
            dir = same_direction(dir, old_dir);
        }

        pos = temp_pos;

        bool not_in_any_range = true;

        for (int j = 0; j < range_count; ++j) {
            auto left_point = range[j].first;
            auto right_point = range[j].second;

            if (left_point.x > right_point.x) {
                auto temp = left_point;
                left_point = right_point;
                right_point = temp;
            }

            if (pos.x >= left_point.x && pos.x <= right_point.x) {
                not_in_any_range = false;
                break;
            }
        }

        if (not_in_any_range) {
            scratches[scratch_index]
                .should_begin_new_line_mask[valid_sample_count] = true;

            continue;
        }

        if (pos.y < center_point.y - half_stroke_width ||
            pos.y > center_point.y + half_stroke_width) {
            scratches[scratch_index]
                .should_begin_new_line_mask[valid_sample_count] = true;

            continue;
        }
        valid_sample_count++;
        if (valid_sample_count >= SAMPLE_POINT_COUNT) {
            printf("Early stop because of too many samples\n");
            break;
        }
    }

    scratches[scratch_index].valid_sample_count = valid_sample_count;

    // if (scratch_index == 0) {
    //     scratches[0].sample_point[0] = center_point;
    //     scratches[0].sample_point[1] = center_point + glm::vec2(0, -1);
    // }
}

void calc_scratches(
    cuda::CUDALinearBufferHandle strokes,
    glm::vec3 camera_position,
    glm::vec3 light_position)
{
    auto stroke_count = strokes->getDesc().element_count;

    unsigned calculation_load = stroke_count * MAX_SCRATCH_COUNT;

    Stroke** d_strokes_ptr =
        reinterpret_cast<Stroke**>(strokes->get_device_ptr());

    GPUParallelFor(
        "calc_scratches", calculation_load, GPU_LAMBDA_Ex(int index) {
            auto related_stroke = index / MAX_SCRATCH_COUNT;
            auto scratch_index = index % MAX_SCRATCH_COUNT;
            auto stroke = d_strokes_ptr[related_stroke];

            stroke->calc_scratch(scratch_index, light_position);
        });
}

void calc_simple_plane_projected_ranges(
    const cuda::CUDALinearBufferHandle& d_strokes,
    glm::vec3 world_camera_position,
    glm::vec2 camera_move_range)
{
    auto stroke_count = d_strokes->getDesc().element_count;
    Stroke** d_strokes_ptr =
        reinterpret_cast<Stroke**>(d_strokes->get_device_ptr());
    GPUParallelFor(
        "calc_simple_projected_ranges", stroke_count, GPU_LAMBDA_Ex(int index) {
            auto stroke = d_strokes_ptr[index];

            auto tangent_vpt =
                stroke->world_to_tangent_point(stroke->virtual_point_position);

            auto camera_left = world_camera_position;
            camera_left.x += camera_move_range.x;

            auto tangent_camera_left =
                stroke->world_to_tangent_point(camera_left);

            auto camera_right = world_camera_position;
            camera_right.x += camera_move_range.y;

            auto tangent_camera_right =
                stroke->world_to_tangent_point(camera_right);

            glm::vec2 on_image_left =
                (tangent_vpt - tangent_camera_left) *
                    (0 - tangent_camera_left.z) /
                    (tangent_vpt.z - tangent_camera_left.z) +
                tangent_camera_left;

            glm::vec2 on_image_right =
                (tangent_vpt - tangent_camera_right) *
                    (0 - tangent_camera_right.z) /
                    (tangent_vpt.z - tangent_camera_right.z) +
                tangent_camera_right;

            if (on_image_left.x > on_image_right.x) {
                auto temp = on_image_left;
                on_image_left = on_image_right;
                on_image_right = temp;
            }

            stroke->range_count = 1;
            stroke->range[0] = std::make_pair(on_image_left, on_image_right);
        });
}
}  // namespace stroke

USTC_CG_NAMESPACE_CLOSE_SCOPE