#include "hip/hip_runtime.h"
#include <glintify/glintify.hpp>

#include "stroke.h"

USTC_CG_NAMESPACE_OPEN_SCOPE
namespace stroke {

// Another question would be how to consider the luminance? the shading?
// By controlling the density of the scratches.
// But how does that mean exactly?

void Stroke::calc_scratch(int scratch_index)
{
    atomicAdd(&scratch_count, 1);
}

void calc_scratches(
    cuda::CUDALinearBufferHandle strokes,
    glm::vec3 camera_position,
    glm::vec3 light_position)
{
    auto stroke_count = strokes->getDesc().element_count;

    unsigned calculation_load = stroke_count * MAX_SCRATCH_COUNT;

    Stroke** d_strokes_ptr =
        reinterpret_cast<Stroke**>(strokes->get_device_ptr());

    GPUParallelFor(
        "calc_scratches", calculation_load, GPU_LAMBDA_Ex(int index) {
            auto related_stroke = index / MAX_SCRATCH_COUNT;
            auto scratch_index = index % MAX_SCRATCH_COUNT;
            auto stroke = d_strokes_ptr[related_stroke];

            stroke->calc_scratch(scratch_index);
        });
}

}  // namespace stroke

USTC_CG_NAMESPACE_CLOSE_SCOPE