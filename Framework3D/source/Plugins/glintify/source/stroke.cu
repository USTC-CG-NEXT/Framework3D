#include "hip/hip_runtime.h"
#include <glintify/glintify.hpp>

#include "stroke.h"

USTC_CG_NAMESPACE_OPEN_SCOPE
namespace stroke {

// Another question would be how to consider the luminance? the shading?
// By controlling the density of the scratches.
// But how does that mean exactly?

__device__ glm::vec3 Stroke::world_to_tangent_point(
    glm::vec3 world)  // A default implementation
{
    return world * 0.5f + glm::vec3(0.5f, 0.5f, 0.0f);
}

__device__ glm::vec3 Stroke::world_to_tangent_vector(glm::vec3 world)
{
    return world;
}

__device__ glm::vec3 Stroke::tangent_to_world_point(glm::vec3 tangent)
{
    return (tangent - glm::vec3(0.5f, 0.5f, 0.0f)) * 2.0f;
}

__device__ glm::vec3 Stroke::tangent_to_world_vector(glm::vec3 tangent)
{
    return tangent;
}

__device__ glm::vec2 Stroke::eval_required_direction(
    glm::vec2 uv_space_pos,
    glm::vec3 light_pos)
{
    auto uv_space_vpt_pos = world_to_tangent_point(virtual_point_position);

    auto tangent_space_cam_dir = uv_space_vpt_pos - glm::vec3(uv_space_pos, 0);

    auto tangent_space_light_dir =
        world_to_tangent_point(light_pos) - glm::vec3(uv_space_pos, 0);

    auto half_vec = 0.5f * (glm::normalize(tangent_space_cam_dir) +
                            glm::normalize(tangent_space_light_dir));

    return -glm::normalize(glm::vec2(-half_vec.y, half_vec.x));
}

__device__ glm::vec2 same_direction(glm::vec2 vec, glm::vec2 reference)
{
    if (glm::dot(vec, reference) < 0) {
        return -vec;
    }
    return vec;
}

__device__ void Stroke::calc_scratch(int scratch_index, glm::vec3 light_pos)
{
    scratch_count = MAX_SCRATCH_COUNT;

    auto left_point = range[0].first;
    auto right_point = range[0].second;

    if (left_point.x > right_point.x) {
        auto temp = left_point;
        left_point = right_point;
        right_point = temp;
    }

    float half_stroke_width = stroke_width / 2.0f;

    unsigned valid_sample_count = 0;

    auto beginner_dir = eval_required_direction(left_point, light_pos);
    auto center_point = left_point + (right_point - left_point) / 2.0f;

    auto ratio = std::abs(beginner_dir.y) / (std::abs(beginner_dir.x) + 0.01f);
    ratio = 1;

    auto case_id = scratch_index % 2;
    auto init_pos_step = scratch_index / 2;

    bool init_pos_going_right = case_id % 2 == 0;

    auto pos = center_point +
               (init_pos_going_right ? glm::vec2(1, 0) : glm::vec2(-1, 0)) *
                   ratio * float(init_pos_step) / float(MAX_SCRATCH_COUNT) /
                   2.0f * (right_point - left_point);

    glm::vec2 old_dir;

    bool allow_going_left = false;

    for (int i = 0; i < SAMPLE_POINT_COUNT; ++i) {
        auto dir = eval_required_direction(pos, light_pos);

        // if (!allow_going_left) {
        //     if (dir.x < 0) {
        //         break;
        //     }
        // }

        auto scratch_going_upward = dir.y > 0;

        if (scratch_going_upward) {
            pos.y -= half_stroke_width;
        }
        else {
            pos.y += half_stroke_width;
        }

        if (i == 0) {
            //if (scratch_going_upward) {
            //    dir *= dir.y > 0 ? 1 : -1;
            //}
            //else {
            //    dir *= dir.y > 0 ? -1 : 1;
            //}
        }
        else {
            dir = same_direction(dir, old_dir);
        }

        old_dir = dir;

        auto step =
            1 / float(SAMPLE_POINT_COUNT) / (std::abs(dir.x) + 0.1f) * 0.5f;

        step = 1;

        scratches[scratch_index].sample_point[i] = pos;
        valid_sample_count++;

        dir = glm::vec2(0, 1);

        pos += dir * step;

        // if (pos.x < left_point.x || pos.x > right_point.x) {
        //     break;
        // }

        // if (pos.y < left_point.y - half_stroke_width ||
        //     pos.y > right_point.y + half_stroke_width) {
        //     allow_going_left = true;
        //     break;
        // }
    }

    scratches[scratch_index].valid_sample_count = valid_sample_count;
}

void calc_scratches(
    cuda::CUDALinearBufferHandle strokes,
    glm::vec3 camera_position,
    glm::vec3 light_position)
{
    auto stroke_count = strokes->getDesc().element_count;

    unsigned calculation_load = stroke_count * MAX_SCRATCH_COUNT;

    Stroke** d_strokes_ptr =
        reinterpret_cast<Stroke**>(strokes->get_device_ptr());

    GPUParallelFor(
        "calc_scratches", calculation_load, GPU_LAMBDA_Ex(int index) {
            auto related_stroke = index / MAX_SCRATCH_COUNT;
            auto scratch_index = index % MAX_SCRATCH_COUNT;
            auto stroke = d_strokes_ptr[related_stroke];

            stroke->calc_scratch(scratch_index, light_position);
        });
}

void calc_simple_plane_projected_ranges(
    const cuda::CUDALinearBufferHandle& d_strokes,
    glm::vec3 world_camera_position,
    glm::vec2 camera_move_range)
{
    auto stroke_count = d_strokes->getDesc().element_count;
    Stroke** d_strokes_ptr =
        reinterpret_cast<Stroke**>(d_strokes->get_device_ptr());
    GPUParallelFor(
        "calc_simple_projected_ranges", stroke_count, GPU_LAMBDA_Ex(int index) {
            auto stroke = d_strokes_ptr[index];

            auto tangent_vpt =
                stroke->world_to_tangent_point(stroke->virtual_point_position);

            auto camera_left = world_camera_position;
            camera_left.x += camera_move_range.x;

            auto tangent_camera_left =
                stroke->world_to_tangent_point(camera_left);

            auto camera_right = world_camera_position;
            camera_right.x += camera_move_range.y;

            auto tangent_camera_right =
                stroke->world_to_tangent_point(camera_right);

            glm::vec2 on_image_left =
                (tangent_vpt - tangent_camera_left) *
                    abs(tangent_camera_left.z) /
                    (tangent_vpt.z - tangent_camera_left.z) +
                tangent_camera_left;

            glm::vec2 on_image_right =
                (tangent_vpt - tangent_camera_right) *
                    abs(tangent_camera_right.z) /
                    (tangent_vpt.z - tangent_camera_right.z) +
                tangent_camera_right;

            stroke->range_count = 1;
            stroke->range[0] = std::make_pair(on_image_left, on_image_right);
        });
}
}  // namespace stroke

USTC_CG_NAMESPACE_CLOSE_SCOPE