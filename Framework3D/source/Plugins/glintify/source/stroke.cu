#include "hip/hip_runtime.h"
#include <glintify/glintify.hpp>

#include "stroke.h"

USTC_CG_NAMESPACE_OPEN_SCOPE
namespace stroke {

// Another question would be how to consider the luminance? the shading?
// By controlling the density of the scratches.
// But how does that mean exactly?

HOST_DEVICE glm::vec3 Stroke::world_to_tangent_point(
    glm::vec3 world)  // A default implementation
{
    return world * 0.5f + glm::vec3(0.5f, 0.5f, 0.0f);
}

HOST_DEVICE glm::vec3 Stroke::world_to_tangent_vector(glm::vec3 world)
{
    return world;
}

HOST_DEVICE glm::vec3 Stroke::tangent_to_world_point(glm::vec3 tangent)
{
    return (tangent - glm::vec3(0.5f, 0.5f, 0.0f)) * 2.0f;
}

HOST_DEVICE glm::vec3 Stroke::tangent_to_world_vector(glm::vec3 tangent)
{
    return tangent;
}

HOST_DEVICE glm::vec2 Stroke::eval_required_direction(
    glm::vec2 uv_space_pos,
    glm::vec3 light_pos)
{
    auto uv_space_vpt_pos = world_to_tangent_point(virtual_point_position);

    glm::vec2 tangent_space_cam_dir =
        uv_space_vpt_pos - glm::vec3(uv_space_pos, 0);
    if (uv_space_vpt_pos.z < 0) {
        tangent_space_cam_dir *= -1;
    }

    glm::vec2 tangent_space_light_dir =
        world_to_tangent_point(light_pos) - glm::vec3(uv_space_pos, 0);

    auto half_vec = 0.5f * (glm::normalize(tangent_space_cam_dir) +
                            glm::normalize(tangent_space_light_dir));

    return glm::normalize(glm::vec2(-half_vec.y, half_vec.x));
}

HOST_DEVICE glm::vec2 same_direction(glm::vec2 vec, glm::vec2 reference)
{
    if (glm::dot(vec, reference) < 0) {
        return -vec;
    }
    return vec;
}

HOST_DEVICE void Stroke::calc_scratch(int scratch_index, glm::vec3 light_pos)
{
    scratch_count = MAX_SCRATCH_COUNT;

    auto left_point = range[0].first;
    auto right_point = range[0].second;

    if (left_point.x > right_point.x) {
        auto temp = left_point;
        left_point = right_point;
        right_point = temp;
    }

    auto tangent_space_light_pos = world_to_tangent_point(light_pos);

    float half_stroke_width = stroke_width / 2.0f;

    unsigned valid_sample_count = 0;

    glm::vec2 center_point;

    center_point.y = left_point.y;

    auto uv_vpt = world_to_tangent_point(virtual_point_position);

    uv_vpt.y = 2.0f * center_point.y - uv_vpt.y;

    glm::vec2 that_direction = uv_vpt - tangent_space_light_pos;
    center_point.x = tangent_space_light_pos.x +
                     (center_point.y - tangent_space_light_pos.y) *
                         that_direction.x / that_direction.y;

    auto pos = center_point + glm::vec2(-1, 0) * float(scratch_index + 0.5f) /
                                  float(MAX_SCRATCH_COUNT) / 1.f;

    glm::vec2 old_dir;

    for (int i = 0; i < SAMPLE_POINT_COUNT; ++i) {
        scratches[scratch_index].should_begin_new_line_mask[i] = false;
    }

    for (int i = 0; i < SAMPLE_POINT_COUNT; ++i) {
        auto dir = eval_required_direction(pos, light_pos);

        if (i == 0) {
            auto scratch_going_right = dir.x > 0;
            if (!scratch_going_right) {
                dir *= -1;
            }

            bool scratch_going_upward = dir.y > 0;
            if (scratch_going_upward) {
                pos.y -= half_stroke_width;
            }
            else {
                pos.y += half_stroke_width;
            }
        }
        else {
            dir = same_direction(dir, old_dir);
        }

        old_dir = dir;

        if (std::abs(dir.y) > 0.999) {
            break;
        }

        auto step = stroke_width / float(SAMPLE_POINT_COUNT) * 50.f;

        pos += dir * step;

        if (pos.x < left_point.x || pos.x > right_point.x) {
            scratches[scratch_index]
                .should_begin_new_line_mask[valid_sample_count] = true;

            continue;
        }

        if (pos.y < left_point.y - half_stroke_width ||
            pos.y > right_point.y + half_stroke_width) {
            scratches[scratch_index]
                .should_begin_new_line_mask[valid_sample_count] = true;
            continue;
        }

        scratches[scratch_index].sample_point[valid_sample_count] = pos;
        valid_sample_count++;
    }

    scratches[scratch_index].valid_sample_count = valid_sample_count;

    // if (scratch_index == 0) {
    //     scratches[0].sample_point[0] = center_point;
    //     scratches[0].sample_point[1] = center_point + glm::vec2(0, -1);
    // }
}

void calc_scratches(
    cuda::CUDALinearBufferHandle strokes,
    glm::vec3 camera_position,
    glm::vec3 light_position)
{
    auto stroke_count = strokes->getDesc().element_count;

    unsigned calculation_load = stroke_count * MAX_SCRATCH_COUNT;

    Stroke** d_strokes_ptr =
        reinterpret_cast<Stroke**>(strokes->get_device_ptr());

    GPUParallelFor(
        "calc_scratches", calculation_load, GPU_LAMBDA_Ex(int index) {
            auto related_stroke = index / MAX_SCRATCH_COUNT;
            auto scratch_index = index % MAX_SCRATCH_COUNT;
            auto stroke = d_strokes_ptr[related_stroke];

            stroke->calc_scratch(scratch_index, light_position);
        });
}

void calc_simple_plane_projected_ranges(
    const cuda::CUDALinearBufferHandle& d_strokes,
    glm::vec3 world_camera_position,
    glm::vec2 camera_move_range)
{
    auto stroke_count = d_strokes->getDesc().element_count;
    Stroke** d_strokes_ptr =
        reinterpret_cast<Stroke**>(d_strokes->get_device_ptr());
    GPUParallelFor(
        "calc_simple_projected_ranges", stroke_count, GPU_LAMBDA_Ex(int index) {
            auto stroke = d_strokes_ptr[index];

            auto tangent_vpt =
                stroke->world_to_tangent_point(stroke->virtual_point_position);

            auto camera_left = world_camera_position;
            camera_left.x += camera_move_range.x;

            auto tangent_camera_left =
                stroke->world_to_tangent_point(camera_left);

            auto camera_right = world_camera_position;
            camera_right.x += camera_move_range.y;

            auto tangent_camera_right =
                stroke->world_to_tangent_point(camera_right);

            glm::vec2 on_image_left =
                (tangent_vpt - tangent_camera_left) *
                    (tangent_camera_left.z) /
                    (tangent_vpt.z - tangent_camera_left.z) +
                tangent_camera_left;

            glm::vec2 on_image_right =
                (tangent_vpt - tangent_camera_right) *
                    (tangent_camera_right.z) /
                    (tangent_vpt.z - tangent_camera_right.z) +
                tangent_camera_right;

            stroke->range_count = 1;
            stroke->range[0] = std::make_pair(on_image_left, on_image_right);
        });
}
}  // namespace stroke

USTC_CG_NAMESPACE_CLOSE_SCOPE